#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void factorablequadratics_kernel(int* x_d, int i) {
    int
}

void factorableQuadratics(int i, int *count) {
    int *count_d;
    *count = 0;

    //Allocate GPU memory
    hipMalloc((void**) &count_d, sizeof(int));

    //Copy data to GPU memory
    hipMemcpy(count_d, count, sizeof(int), hipMemcpyHostToDevice);

    //Start time
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Perform computation on GPU
    int numThreadsPerBlock = 512;
    int numBlocks = (2 * i * i * i + numThreadsPerBlock - 1) / numThreadsPerBlock);
    factorablequadratics_kernel<<<numBlocks, numThreadsPerBlock>>>(count_id, i);

    //Synchronize
    hipDeviceSynchronize();

    //End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);

    //Copy data from GPU memory
    hipMemcpy(count, count_d, sizeof(int), hipMemcpyDeviceToHost);

    //Deallocate GPU memory
    hipFree(count_d);
}


int main() {
    int i = 10;
    int *count = (int*)malloc(sizeof(int));

    //PERFORM FACTORABLE QUADRATICS
    factorableQuadratics(i, count);

    //FREE
    free(count);

    return 0;
}