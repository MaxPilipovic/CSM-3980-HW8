
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void factorablequadratics_kernel(int* count, int i) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int start = index; start < 2 * i * i * i; start += stride) {
        int a = (start / ((2 * i + 1) * (2 * i + 1)));
        printf("A VALUE %d\n", a);
        int b = (start / ((2 * i + 1) % (2 * i + 1)));
        printf("B VALUE %d\n", b);
        int c = (start % (2 * i + 1));
        printf("C VALUE %d\n", c);

        if (a != 0 && b != 0 && c != 0) {
            int check = b * b - 4 * a * c;
            if (check >= 0) {
                int squareRoot = (int)sqrtf(check);
                if (squareRoot * squareRoot == check) {
                    atomicAdd(count, 1);
                }
            }
        }
    }
}

void factorableQuadratics(int i, int *count) {
    int *count_d;
    *count = 0;

    //Allocate GPU memory
    hipMalloc((void**) &count_d, sizeof(int));

    //Copy data to GPU memory
    hipMemcpy(count_d, count, sizeof(int), hipMemcpyHostToDevice);

    //Start time
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Perform computation on GPU
    int numThreadsPerBlock = 512;
    int numBlocks = (2 * i * i * i + numThreadsPerBlock - 1) / numThreadsPerBlock;
    factorablequadratics_kernel<<<numBlocks, numThreadsPerBlock>>>(count_d, i);

    //Synchronize
    hipDeviceSynchronize();

    //End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);

    //Copy data from GPU memory
    hipMemcpy(count, count_d, sizeof(int), hipMemcpyDeviceToHost);

    //Deallocate GPU memory
    hipFree(count_d);
}

int main() {
    int i = 2;
    int *count = (int*)malloc(sizeof(int));

    //PERFORM FACTORABLE QUADRATICS
    factorableQuadratics(i, count);

    //PRINT
    printf("%d\n", *count);

    //FREE
    free(count);

    return 0;
}
