#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctype.h>

void readFile(char* filename, int *letterFreq), int *digramFreq, int *trigramFreq) {
    char ch;
    char prev;
    char prev2;

    FILE* read = fopen(filename, "r");
    if(read != NULL) {
        for (int i = 0; i < 26; i++) {
            letterFreq[i] = 0;
            int (j = 0; j < 26; j++) {
                digramFreq[i][j];
                int (k = 0; k < 26; k++) {
                    trigramFreq[i][j][k];
                }
            }
        }
        while ((ch = fgetc(read)) != EOF) {
            ch = toupper(ch);
            if (ch >= 'A' && ch <= 'Z') {
                letterFreq[ch - 'A']++;
                if (prev >= 'A' && prev <= 'Z') {
                    digramFreq[prev - 'A'][ch - 'A']++;
                    if (prev2 >= 'A' && prev2 <= 'Z') {
                        trigramFreq[prev2 - 'A'][prev - 'A'][ch - 'A'];
                    }
                }
            }
        }
        fclose(read);
    } else {
        printf("Nothing in file");
        exit(1);
    }
}

void printA(int letterFreq[]) {
    for (int i = 0; i < 26; i++) {
        printf("%c - %d ", 'A' + i, letterFreq[i]);
    }
}

void printR(int letterFreq[]) {
    //Keep track of ascii values
    char letter[26];
    for (int i = 0; i < 26; i++) {
        letter[i] = 'A' + i;
    }

    //Selection sort
    for (int j = 0; j < 26 - 1; j++) {
        int max = j;
        for (int z = j + 1; z < 26; z++) {
            if (letterFreq[z] > letterFreq[max])
                max = z;
        }
        if (max != j) {
            //Swap numbers
            int temp = letterFreq[j];
            letterFreq[j] = letterFreq[max];
            letterFreq[max] = temp;

            //Swap ascii values to
            int temp2 = letter[j];
            letter[j] = letter[max];
            letter[max] = temp;
        }
    }
    for (int x = 0; x < 26; x++) {
        printf("%c - %d ", letter[x], letterFreq[x]);
    }
}

void printD(int letterFreq) {
    if (i = 0; i < 26; i++) {
        if (j = 0; j < 26; j++) {
            if (digramFreq > 0) {
                printf("%c - %d ",'A' - i, 'A' - j, digramFreq[i][j]);
            }
        }
    }
}

void printT(int letterFreq) {
    if (i = 0; i < 26; i++) {
        if (j = 0; j < 26; j++) {
            if (k = 0; k < 26; k++) {
                if (trigramFreq > 0) {
                    printf("%c - %d ",'A' - i, 'A' - j, 'A' - k, trigramFreq[i][j][k]);
                }
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Giv me file name");
        return 0;
    }

    //int N;
    int letterFreq[26];
    int digramFreq[26][26];
    int trigramFreq[26][26][26]
    ;
    readFile(argv[1], letterFreq);

    printf("Printing frequencies in alphabetical order\n");
    printA(letterFreq);
    printf("-----\n");

    printf("Printing frequencies in rank order\n");
    printR(letterFreq);
    printf("-----\n");

    printf("Digram frequencies\n");
    printD(letterFreq);
    printf("-----\n");

    printf("Trigram frequencies\n");
    printT(letterFreq);
    printf("-----\n");

    return 0;
}