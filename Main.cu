
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctype.h>
void textfrequencycounter(unsigned char* hostData, int size, int* letterFreq, int digramFreq[26][26], int trigramFreq[26][26][26]) {
    for (int i = 0; i < 26; i++) {
        letterFreq[i] = 0;
        for (int j = 0; j < 26; j++) {
            digramFreq[i][j] = 0;
            for (int k = 0; k < 26; k++) {
                trigramFreq[i][j][k] = 0;
            }
        }
    }

    for (int i = 0; i < size; i++) {
        if (isalpha(hostData[i])) {
            int value = tolower(hostData[i]) - 'a';
            letterFreq[value]++;

            if (i + 1 < size && isalpha(hostData[i + 1])) {
                int value2 = tolower(hostData[i + 1]) - 'a';
                digramFreq[value][value2]++;
            }

            if (i + 2 < size && isalpha(hostData[i + 2])) {
                int value2 = tolower(hostData[i + 1]) - 'a';
                int value3 = tolower(hostData[i + 2]) - 'a';
                trigramFreq[value][value2][value3]++;
            }
        }
    }
}


 int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Giv me file name");
        return 1;
    }

     // Open file
     FILE *file = fopen(argv[1], "r");
     if(!file) {
         fprintf(stderr, "Unable to open %s!\n", argv[1]);
         return 2;
     }

     //Find file size
     fseek(file, 0, SEEK_END);
     long size = ftell(file);
     fseek(file, 0, SEEK_SET);

     // Arrays to hold our data on the host and gpu
     unsigned char *hostData = (unsigned char*) malloc((size + 1) * sizeof(char));

     //Read
     printf("Reading %s\n", argv[1]);
     fread(hostData, size, 1, file);
     fclose(file);
     hostData[size] = '\0';

     //Hold Data
     int letterFreq[26];
     int digramFreq[26][26];
     int trigramFreq[26][26][26];

     textfrequencycounter(hostData, size, letterFreq, digramFreq, trigramFreq);

     //Print letter frequencies
     for(int i = 0; i < 26; i++) {
         printf("%c : %5d : %5d : %5d\n",
                (char)('A' + i),
                letterFreq[i]);
     }

     //Print digram frequencies
     for (int i = 0; i < 26; i++) {
         for (int j = 0; j < 26; j++) {
             if (digramFreq[i][j] > 0) {
                 printf("%c : %5d : %5d : %5d\n",
                        (char)('A' + i),
                        (char)('A' + j),
                        digramFreq[i][j]);
             }
         }
     }

     //Print trigram frequencies
     for (int i = 0; i < 26; i++) {
         for (int j = 0; j < 26; j++) {
             for (int k = 0; k < 26; k++) {
                 if (trigramFreq[i][j][k] > 0) {
                     printf("%c : %5d : %5d : %5d\n",
                            (char)('A' + i),
                            (char)('A' + j),
                            (char)('A' + k),
                            trigramFreq[i][j][k]);
                 }
             }
         }
     }
}