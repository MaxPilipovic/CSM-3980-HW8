
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctype.h>

void readFile(char* filename, int *freqLetters) {
    char ch;

    FILE* read = fopen(filename, "r");
    if(read != NULL) {
        for (int i = 0; i < 26; i++) {
            freqLetters[i] = 0;
        }
        while ((ch = fgetc(read)) != EOF) {
            ch = toupper(ch);
            if (ch >= 'A' && ch <= 'Z') {
                freqLetters[ch - 'A']++;
            }
        }
        fclose(read);
    } else {
        printf("Nothing in file");
        exit(1);
    }
}

void printA(int freqLetters[]) {
    for (int i = 0; i < 26; i++) {
        printf("%c - %d ", 'A' + i, freqLetters[i]);
    }
}

void printR(int letterFreq[]) {
    //Keep track of ascii values
    char letter[26];
    for (int i = 0; i < 26; i++) {
        letter[i] = 'A' + i;
    }

    //Selection sort
    for (int j = 0; j < 26 - 1; j++) {
        int max = j;
        for (int z = j + 1; z < 26; z++) {
            if (letterFreq[z] > letterFreq[max])
                max = z;
        }
        if (max != j) {
            //Swap numbers
            int temp = letterFreq[j];
            letterFreq[j] = letterFreq[max];
            letterFreq[max] = temp;

            //Swap ascii values to
            int temp2 = letter[j];
            letter[j] = letter[max];
            letter[max] = temp;
        }
    }
    for (int x = 0; x < 26; x++) {
        printf("%c - %d ", letter[x], letterFreq[x]);
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Giv me file name");
        return 0;
    }

    //int N;
    int letterFreq[26];
    readFile(argv[1], letterFreq);

    printf("Printing frequencies in alphabetical order\n");
    printA(letterFreq);

    printf("\n");

    printf("Printing frequencies in rank order\n");
    printR(letterFreq);

    return 0;
}