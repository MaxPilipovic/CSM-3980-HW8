#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctype.h>

void readFile(char* filename, int *freqLetters) {
    char ch;

    FILE* read = fopen(filename, "r");
    if(read != NULL) {
        for (int i = 0; i < 26; i++) {
            freqLetters[i] = 0;
        }
        while ((ch = fgetc(read)) != EOF) {
            ch = toupper(ch);
            if (ch >= 'A' && ch <= 'Z') {
                freqLetters[ch - 'A']++;
            }
        }
        fclose(read);
    } else {
        printf("Nothing in file");
        exit(1);
    }
}

void printA(int freqLetters[]) {
    for (int i = 0; i < 26; i++) {
        printf("%c - %d ", 'A' + i, freqLetters[i]);
    }
}

void printR(int letterFreq[]) {
    for (int i = 0; i < 26 - 1; i++) {
        int max = i;
        for (int j = i + 1; j < 26; j++) {
            if (letterFreq[j] > letterFreq[min])
                max = j;
        }
        if (max != i) {
            letterFreq[max] = letterFreq[i]);
        }
    }
    for (int x = 0; x < 26; x++) {
        printf("%c - %d ", 'A' + x, freqLetters[x]);
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Giv me file name");
        return 0;
    }

    //int N;
    int letterFreq[26];
    readFile(argv[1], letterFreq);

    printf("Printing frequencies in alphabetical order\n");
    printA(letterFreq);

    printf("\n");

    printf("Printing frequencies in rank order\n");
    printR(letterFreq);

    return 0;
}