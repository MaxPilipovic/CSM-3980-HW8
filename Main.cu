
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctype.h>

void readFile(char* filename, int *freqLetters) {
    char ch;

    FILE* read = fopen(filename, "r");
    if(read != NULL) {
        for (int i = 0; i < 26; i++) {
            freqLetters[i] = 0;
        }
        while ((ch = fgetc(read)) != EOF) {
            ch = toupper(ch);
            if (ch >= 'A' && ch <= 'Z') {
                freqLetters[ch - 'A']++;
            }
        }
        fclose(read);
    } else {
        printf("Nothing in file");
        exit(1);
    }
}

void printA(int freqLetters[]) {
    for (int i = 0; i < 26; i++) {
        printf("%d ", i, " ", freqLetters[i]);
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Giv me file name");
        return 0;
    }

    //int N;
    int letterFreq[26];
    readFile(argv[1], letterFreq);
    printf("Printing frequencies in alphabetical order\n");
    printA(letterFreq);

    return 0;
}