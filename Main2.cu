#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void digram_textfrequencycounter_kernel(unsigned char* s, int* result, int n) {
    //Private copies of the result for each block
    __shared__ int privateResult[676];

    //Initialize private result
    if (threadIdx.x < 676) {
        privateResult[threadIdx.x] = 0;
    }

    __syncthreads();

    //Stride
    //Get starting location and stride. Striding this way gives
    //coalesced memory access
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < n - 1) {
        //Handle uppercase
        int value = s[i] - 'a';
        int value2 = s[i + 1] - 'a';
        if (value >= 0 && value < 26 && value2 >= 0 && value2 < 26) {
            atomicAdd(&(privateResult[value*26 + value2]), 1);
        }
        __syncthreads();

        //Store final result
        if (threadIdx.x < 676) {
            atomicAdd(&(result[threadIdx.x]), privateResult[threadIdx.x]);
        }
    }
}

__global__ void trigram_textfrequencycounter_kernel(unsigned char* s, int* result, int n) {
    //Private copies of the result for each block
    __shared__ int privateResult[17576];

    //Initialize private result
    if(threadIdx.x < 17576) {
        privateResult[threadIdx.x] = 0;
    }

    __syncthreads();

    //Get starting location and stride. Striding this way gives
    //coalesced memory access
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while(i < n - 2) {
        //Handle uppercase
        int value = s[i] - 'a';
        int value2 = s[i + 1] - 'a';
        int value3 = s[i + 2] - 'a';
        if(value >= 0 && value < 26 && value2 >= 0 && value2 < 26 && value3 >= 0 && value3 < 26) {
            atomicAdd(&(privateResult[value*26*26 + value2*26 + value3]), 1);
        }
        __syncthreads();

        //Store final result
        if(threadIdx.x < 17576) {
            atomicAdd(&(result[threadIdx.x]), privateResult[threadIdx.x]);
        }
    }

__global__ void textfrequencycounter_kernel(unsigned char* s, int* result, int n) {
    //Private copies of the result for each block
    __shared__ int privateResult[26];

    //Initialize private result
    if(threadIdx.x < 26) {
        privateResult[threadIdx.x] = 0;
    }

    __syncthreads();

    //Get starting location and stride. Striding this way gives
    //coalesced memory access
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while(i < n) {
        //Handle uppercase
        int resultIndex = s[i] - 'A';
        if(resultIndex >= 0 && resultIndex < 26) {
            atomicAdd(&(privateResult[resultIndex]), 1);
        }

        //Handle lowercase
        resultIndex = s[i] - 'a';
        if(resultIndex >= 0 && resultIndex < 26) {
            atomicAdd(&(privateResult[resultIndex]), 1);
        }
        i+=stride;
    }

    __syncthreads();

    //Store final result
    if(threadIdx.x < 26) {
        atomicAdd(&(result[threadIdx.x]), privateResult[threadIdx.x]);
    }
}

int main(int argc, const char* argv[]) {
    //Check that a file name was given
    if(argc < 2) {
        printf("Giv me file name");
        return 1;
    }

    //Open file
    FILE *file = fopen(argv[1], "r");
    if(!file) {
        fprintf(stderr, "Unable to open %s!\n", argv[1]);
        return 2;
    }

    //Find file size
    fseek(file, 0, SEEK_END);
    long size = ftell(file);
    fseek(file, 0, SEEK_SET);

    //Arrays to hold our data on the host and gpu
    unsigned char *hostData = (unsigned char*) malloc((size + 1) * sizeof(char));
    unsigned char *gpuData;
    hipMalloc(&gpuData, (size + 1) * sizeof(char));

    //Read data (adding null)
    printf("Reading %s\n", argv[1]);
    fread(hostData, size, 1, file);
    fclose(file);
    hostData[size] = '\0';

    //Allocate arrays to hold our results
    int *result;
    hipMallocManaged(&result, 26*sizeof(int));

    int *digram;
    int *trigram;

    hipMallocManaged(&digram, 676*sizeof(int));
    hipMallocManaged(&trigram, 17576*sizeof(int));

    //Set the size of our grid. Threads must be >= 256.
    int threads = 256;
    int blocks = 80;

    //Copy data to GPU. I am not sure if it is worth keeping the
    //null at the end of the text as it is not used as a terminator
    //in our kernels, but I was taught to always terminate my strings...
    hipMemcpy(gpuData, hostData, (size + 1) * sizeof(char), hipMemcpyHostToDevice);
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(result, 26 * sizeof(int), device);
    hipMemPrefetchAsync(digram, 676 * sizeof(int), device);
    hipMemPrefetchAsync(trigram, 15576 * sizeof(int), device);

    //Start time
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Run histograms
    textfrequencycounter_kernel<<<blocks,threads>>>(gpuData, result, size);

    digram_textfrequencycounter_kernel<<<blocks,threads>>>(gpuData, digram, size);

    trigram_textfrequencycounter_kernel<<<blocks,threads>>>(gpuData, trigram, size);

    //End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);

    //Move results back to host
    hipMemPrefetchAsync(result, 26 * sizeof(int), hipCpuDeviceId);
    hipMemPrefetchAsync(digram, 676 * sizeof(int), hipCpuDeviceId);
    hipMemPrefetchAsync(trigram, 15576 * sizeof(int), hipCpuDeviceId);

    hipDeviceSynchronize();

    //Print results
    for(int index = 0; index < 26; index++) {
        printf("%c : %5d : %5d : %5d\n",
               (char)('A' + index),
               result[index]);
    }

    //Clean up
    free(hostData);
    hipFree(gpuData);
    hipFree(result);
    hipFree(digram);
    hipFree(trigram);
    return 0;
}
