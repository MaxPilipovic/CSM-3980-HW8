
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void textfrequencycounter_kernel() {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

}

void textfrequencycounter() {

    //Allocate GPU memory

    //Copy data to GPU memory


    //Start time
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Perform computation on GPU

    //Synchronize
    hipDeviceSynchronize();

    //End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);

    //Copy data from GPU memory


    //Deallocate GPU memory

}


int main() {

    //PERFORM TEXT FREQUENCY COUNTER

    //FREE


    return 0;
}