#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void textfrequencycounter_kernel(unsigned char* s, int* result, int n) {
    //Private copies of the result for each block
    __shared__ int privateResult[26];

    //Initialize private result
    if(threadIdx.x < 26) {
        privateResult[threadIdx.x] = 0;
    }

    __syncthreads();

    //Get starting location and stride. Striding this way gives
    //coalesced memory access
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while(i < n) {
        //Handle uppercase
        int resultIndex = s[i] - 'A';
        if(resultIndex >= 0 && resultIndex < 26) {
            atomicAdd(&(privateResult[resultIndex]), 1);
        }

        //Handle lowercase
        resultIndex = s[i] - 'a';
        if(resultIndex >= 0 && resultIndex < 26) {
            atomicAdd(&(privateResult[resultIndex]), 1);
        }
        i+=stride;
    }

    __syncthreads();

    //Store final result
    if(threadIdx.x < 26) {
        atomicAdd(&(result[threadIdx.x]), privateResult[threadIdx.x]);
    }
}

int main(int argc, const char* argv[]) {
    //Check that a file name was given
    if(argc < 2) {
        printf("Giv me file name");
        return 1;
    }

    //Open file
    FILE *file = fopen(argv[1], "r");
    if(!file) {
        fprintf(stderr, "Unable to open %s!\n", argv[1]);
        return 2;
    }

    //Find file size
    fseek(file, 0, SEEK_END);
    long size = ftell(file);
    fseek(file, 0, SEEK_SET);

    //Arrays to hold our data on the host and gpu
    unsigned char *hostData = (unsigned char*) malloc((size + 1) * sizeof(char));
    unsigned char *gpuData;
    hipMalloc(&gpuData, (size + 1) * sizeof(char));

    //Read data (adding null)
    printf("Reading %s\n", argv[1]);
    fread(hostData, size, 1, file);
    fclose(file);
    hostData[size] = '\0';

    //Allocate arrays to hold our results
    int *result;
    hipMallocManaged(&result, 26*sizeof(int));

    //Set the size of our grid. Threads must be >= 256.
    int threads = 256;
    int blocks = 80;

    //Copy data to GPU. I am not sure if it is worth keeping the
    //null at the end of the text as it is not used as a terminator
    //in our kernels, but I was taught to always terminate my strings...
    hipMemcpy(gpuData, hostData, (size + 1) * sizeof(char), hipMemcpyHostToDevice);
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(result, 26 * sizeof(int), device);

    //Run histograms
    textfrequencycounter_kernel<<<blocks,threads>>>(gpuData, result, size);

    //Move results back to host
    hipMemPrefetchAsync(result, 26 * sizeof(int), hipCpuDeviceId);

    hipDeviceSynchronize();

    //Print results
    for(int index = 0; index < 26; index++) {
        printf("%c : %5d : %5d : %5d\n",
               (char)('A' + index),
               result[index]);
    }

    //Clean up
    free(hostData);
    hipFree(gpuData);
    hipFree(result);

    return 0;
}
